#include "texture_array.h"

TextureArray::~TextureArray()
{
  if (num_images > 0) {
    for (int i = 0; i < num_images; ++i) {
      checkCudaErrors(hipDestroyTextureObject(data[i]));
      checkCudaErrors(hipFreeArray(arrays[i]));
    }
  }
}

void TextureArray::setDataToTextureMemory(const std::vector<Mat2D>& mat)
{
  num_images = mat.size();

  for (int i = 0; i < num_images; i++) {
    const int rows = mat[i].height;
    const int cols = mat[i].width;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors(hipMallocArray(&arrays[i], &channelDesc, cols, rows));
    checkCudaErrors(hipMemcpy2DToArray(
        arrays[i], 0, 0, mat[i].data.ptr(), mat[i].data.step[0], cols * sizeof(float), rows, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arrays[i];

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    checkCudaErrors(hipCreateTextureObject(&(data[i]), &resDesc, &texDesc, nullptr));
  }
}